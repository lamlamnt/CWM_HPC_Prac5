#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

float calculate_mean(float *input,int n){
	float sum = 0;
	for(int i = 0; i < n; i++){
		sum +=input[i];
	}
	return float(sum/float(n));
}

float calculate_std(float *input, float mean, int n){
	float sum_squared = 0;
	for(int i = 0; i<n; i++){
		sum_squared += (input[i] - mean)*(input[i] - mean);
	}	
	return float(sqrt(sum_squared/float(n)));
}

//Output histogram to a file
//Histogram divided into less than -1, between -1 and -0.5, between -0.5 and 0, between 0 and 0.5, between 0.5 and 1, and greater
//than 1
void generate_histogram(float *input, int n){
	/*
	int count_1; int count_2; int count_3, int count_4, int count_5, int count_6;
	for(int i = 0; i < n; i++){
		if(input[i] < -1) count_1 +=1;
		else if(input[i] > -1 && input[i] < -0.5) {count_2 +=1};
		else if(input[i] > -0.5 && input[i] < 0) {count_3 +=1};
		else if(input[i] > 0 && input[i] < 0.5) {count_4 +=1};
		else if(input[i] > 0.5 && input[i] < 1) {count_5 +=1};
		else{count_6 +=1;}
	}
	*/
	//Writes the input values to a file
	FILE *fp;
	fp = fopen("data.dat","w");
	for(int i = 0; i < n; i++){
		fprintf(fp,"%f\n",input[i]);
	}
	fclose(fp);
}

int main(void){
	//Allocate pointers for host and device memory
	float *h_output;
	float *d_output;
	int n = 10;
	int mem_size = n*sizeof(float);

	h_output = (float*)malloc(mem_size);

	hipMalloc((void**)&d_output,mem_size);

	//Declare variable
	hiprandGenerator_t gen;

	//Create random number generator
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);

	//Set the generator options
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

	//Generate the randoms
	hiprandGenerateNormal(gen, d_output,n, 0.0f, 1.0f);

	//Copy result from device to host
	hipMemcpy(h_output,d_output,mem_size,hipMemcpyDeviceToHost);

	float mean = calculate_mean(h_output,n);
	printf("The mean is %f \n",mean);
		
	float std = calculate_std(h_output,mean,n);
	printf("The standard deviation is %f \n",std);
	
	//Output histogram to a png
	generate_histogram(h_output,n);
	//cleanup memory
	free(h_output);
	hipFree(d_output);
	return(0);
}
